/**************************************************
# Copyright (C) 2014 Raptis Dimos <raptis.dimos@yahoo.gr>
# This program is free software: you can redistribute it and/or modify
# it under the terms of the GNU General Public License as published by
# the Free Software Foundation, either version 3 of the License, or
# (at your option) any later version.
# This program is distributed in the hope that it will be useful,
# but WITHOUT ANY WARRANTY; without even the implied warranty of
# MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
# GNU General Public License for more details.
# You should have received a copy of the GNU General Public License
# along with this program. If not, see <http://www.gnu.org/licenses/>.
# **************************************************/

/*
 *  gpu_util.cu -- GPU utility functions
 *  Copyright (C) 2010-2013, Computing Systems Laboratory (CSLab)
 */ 

#include <hip/hip_runtime.h>
#include "gpu_util.h"

/* Initialize the CUDA runtime */
void gpu_init()
{
    hipFree(0);
}

void *gpu_alloc(size_t count)
{
    void *ret;
    if (hipMalloc(&ret, count) != hipSuccess) {
        ret = NULL;
    }

    return ret;
}

void gpu_free(void *gpuptr)
{
    hipFree(gpuptr);
}

int copy_to_gpu(const void *host, void *gpu, size_t count)
{
    if (hipMemcpy(gpu, host, count, hipMemcpyHostToDevice) != hipSuccess)
        return -1;
    return 0;
}

int copy_from_gpu(void *host, const void *gpu, size_t count)
{
    if (hipMemcpy(host, gpu, count, hipMemcpyDeviceToHost) != hipSuccess)
        return -1;
    return 0;
}

const char *gpu_get_errmsg(hipError_t err)
{
    return hipGetErrorString(err);
}

const char *gpu_get_last_errmsg()
{
    return gpu_get_errmsg(hipGetLastError());
}
    
