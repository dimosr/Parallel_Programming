#include "hip/hip_runtime.h"
/*
 *  gpu_kernels.cu -- GPU kernels
 *
 *  Copyright (C) 2014, Computing Systems Laboratory (CSLab)
 *  Copyright (C) 2014, Vasileios Karakasis
 */ 

#include <stdio.h>
#include <hip/hip_runtime.h>
#include "error.h"
#include "gpu_util.h"
#include "graph.h"
#include "timer.h"
#include <math.h>

#define GPU_KERNEL_NAME(name)   do_apsp_gpu ## name

weight_t *copy_graph_to_gpu(const graph_t *graph)
{
    size_t dist_size = graph->nr_vertices*graph->nr_vertices;
    weight_t *dist_gpu = (weight_t *) gpu_alloc(dist_size*sizeof(*dist_gpu));
    if (!dist_gpu)
        error(0, "gpu_alloc() failed: %s", gpu_get_last_errmsg());

    if (copy_to_gpu(graph->weights[0], dist_gpu,
                    dist_size*sizeof(*dist_gpu)) < 0)
        error(0, "copy_to_gpu() failed: %s", gpu_get_last_errmsg());

    return dist_gpu;
}

graph_t *copy_graph_from_gpu(const weight_t *dist_gpu, graph_t *graph)
{
    size_t dist_size = graph->nr_vertices*graph->nr_vertices;

    if (copy_from_gpu(graph->weights[0], dist_gpu,
                      dist_size*sizeof(*dist_gpu)) < 0)
        error(0, "copy_from_gpu() failed: %s", gpu_get_last_errmsg());

    return graph;
}

/*
 *  The naive GPU kernel
 */ 
__global__ void GPU_KERNEL_NAME(_naive)(weight_t *dist, int n, int k)
{
	int i_equal,j_equal;
    // FILLME: the naive GPU kernel code
	
	/* if Grid & blocks are 1-dimensional */
	//i_equal = (blockIdx.x*blockDim.x + threadIdx.x) / n;
        //j_equal = (blockIdx.x*blockDim.x + threadIdx.x) % n;
	//check 266
	/* if Grid & block  are 2-dimensional */
	i_equal = blockIdx.y*blockDim.y + threadIdx.y;
	j_equal = blockIdx.x*blockDim.x + threadIdx.x;

        if (dist[(i_equal*n+j_equal)] > dist[(i_equal*n+k)] + dist[(n*k+j_equal)]){
        	dist[(i_equal*n+j_equal)] = dist[(i_equal*n+k) ] + dist[(n*k+j_equal)];
        }
   
}


/*
 *  The tiled GPU kernel(s) using global memory
 */ 
__global__ void GPU_KERNEL_NAME(_tiled_stage_1)(weight_t *dist, int n,
                                                int k_tile, int tile_dim)
{
    int i,j,inner_k;          //inner_k used for iterating inside the tile
    

    int i_equal = threadIdx.y;
    int j_equal = threadIdx.x;      //i,j indexes inside the tile 
    i = k_tile*blockDim.y + threadIdx.y;
    j = k_tile*blockDim.x + threadIdx.x;	    //i,j in the whole array
    int index = i*n + j;				   //index in the 1D array 
    for(inner_k=0; inner_k<tile_dim; inner_k++){
            if( dist[index] > dist[index - (j_equal - inner_k)] + dist[index - (i_equal - inner_k)*n] ){
               	dist[index] = dist[index - (j_equal - inner_k)] + dist[index - (i_equal - inner_k)*n];
            }
            __syncthreads();        
    }
	
}
    

__global__ void GPU_KERNEL_NAME(_tiled_stage_2)(weight_t *dist, int n,
                                                int k_tile, int tile_dim)
{
    int i,j,inner_k;             //inner_k used for iterating inside the tile
    
    if( (blockIdx.y == 0) && (blockIdx.x != k_tile) ){//if the running block belongs to the vertical line (1st semi-half of grid)  and not in the (k,k) tile
            int i_equal = threadIdx.y;
    	    int j_equal = threadIdx.x;
	    i = blockIdx.x*blockDim.y + threadIdx.y;
	    j = k_tile*blockDim.x + threadIdx.x;
	    int index = i*n + j;
	    for(inner_k=0; inner_k<tile_dim; inner_k++){        
                if( dist[index] > dist[index - (j_equal - inner_k)] + dist[index - (i_equal - inner_k)*n - (blockIdx.x - k_tile)*blockDim.x*n]){
                    dist[index] = dist[index - (j_equal - inner_k)] + dist[index - (i_equal - inner_k)*n - (blockIdx.x - k_tile)*blockDim.x*n];
                }
                __syncthreads();
            }
    }
    if( (blockIdx.y == 1) && (blockIdx.x != k_tile) ){     //if the running block belongs to the horizontal line (2nd semi-half of grid)  and not in the (k,k) tile
     	int i_equal = threadIdx.y;
    	int j_equal = threadIdx.x;
	i = k_tile*blockDim.y + threadIdx.y;
        j = blockIdx.x*blockDim.x + threadIdx.x;
        int index = i*n + j;
    	for(inner_k=0; inner_k<tile_dim; inner_k++){
          	if( dist[index] > dist[index - (j_equal - inner_k) - (blockIdx.x - k_tile)*blockDim.x] + dist[index - (i_equal - inner_k)*n] ){
                       dist[index] = dist[index - (j_equal - inner_k) - (blockIdx.x - k_tile)*blockDim.x] + dist[index - (i_equal - inner_k)*n];
          	}
          	__syncthreads();
        }
    }
             
}

__global__ void GPU_KERNEL_NAME(_tiled_stage_3)(weight_t *dist, int n,
                                                int k_tile, int tile_dim)
{
    int i,j,inner_k;             //inner_k used for iterating inside the tile
    
    
    int i_equal = threadIdx.y;
    int j_equal = threadIdx.x;
    i = blockIdx.y*blockDim.y + threadIdx.y;
    j = blockIdx.x*blockDim.x + threadIdx.x;
    int index = i*n + j;	
    if( (blockIdx.x != k_tile) && (blockIdx.y != k_tile) ){       //this will be executed only by the blocks not belonging to the k_tile 
        for(inner_k=0; inner_k<tile_dim; inner_k++){
            if( dist[index] > dist[index - (j_equal - inner_k) - (blockIdx.x - k_tile)*blockDim.x] + dist[index - (i_equal - inner_k)*n - ( blockIdx.y - k_tile)*blockDim.y*n] ){
                dist[index] = dist[index - (j_equal - inner_k) - (blockIdx.x - k_tile )*blockDim.x] + dist[index - (i_equal - inner_k)*n - ( blockIdx.y - k_tile)*blockDim.y*n];
            }
        }
    }

}

__global__ void GPU_KERNEL_NAME(_tiled_shmem_stage_1)(weight_t *dist, int n,
                                                      int k_tile, int tile_dim)
{
    int i,j,i_equal,j_equal,inner_k;

    __shared__ weight_t local_tile[GPU_TILE_DIM][GPU_TILE_DIM];

    i_equal = threadIdx.y;      
    j_equal = threadIdx.x;      //i_equal,j_equal indexes inside the tile
    i = k_tile*blockDim.y + threadIdx.y;
    j = k_tile*blockDim.x + threadIdx.x;        //i,j indexes in the whole array
    int index = i*n + j;            //index in the 1D array
    local_tile[i_equal][j_equal] = dist[index]; 
    __syncthreads();
    for(inner_k = 0; inner_k < tile_dim; inner_k++){
            if( local_tile[i_equal][j_equal] > local_tile[i_equal][inner_k] + local_tile[inner_k][j_equal] ){
                local_tile[i_equal][j_equal] = local_tile[i_equal][inner_k] + local_tile[inner_k][j_equal];
            }
            __syncthreads();
    }
    dist[index] = local_tile[i_equal][j_equal];

}

__global__ void GPU_KERNEL_NAME(_tiled_shmem_stage_2)(weight_t *dist, int n,
                                                      int k_tile, int tile_dim)
{
    int i,j,i_equal,j_equal,inner_k;

    __shared__ weight_t local_KK_tile[GPU_TILE_DIM][GPU_TILE_DIM];
    __shared__ weight_t local_IK_tile[GPU_TILE_DIM][GPU_TILE_DIM];
    __shared__ weight_t local_KI_tile[GPU_TILE_DIM][GPU_TILE_DIM];

    if( (blockIdx.y == 0) && (blockIdx.x != k_tile) ){//if the running block belongs to the vertical line (1st semi-half of grid) and not in the (k,k) tile
	int i_equal = threadIdx.y;
        int j_equal = threadIdx.x;
        i = blockIdx.x*blockDim.y + threadIdx.y;
        j = k_tile*blockDim.x + threadIdx.x;
        int index = i*n + j;
        local_KK_tile[i_equal][j_equal] = dist[index - (blockIdx.x - k_tile)*blockDim.x*n];
        local_IK_tile[i_equal][j_equal] = dist[index];
	__syncthreads();
	    
        for(inner_k=0; inner_k<tile_dim; inner_k++){
            if( local_IK_tile[i_equal][j_equal] > local_IK_tile[i_equal][inner_k] + local_KK_tile[inner_k][j_equal] ){
                local_IK_tile[i_equal][j_equal] = local_IK_tile[i_equal][inner_k] + local_KK_tile[inner_k][j_equal];
            }
            __syncthreads();
        }
        dist[index] = local_IK_tile[i_equal][j_equal];
    }
    if( (blockIdx.y == 1) && (blockIdx.x != k_tile)  ){     //if the running block belongs to the horizontal line and not in the (k,k) tile
	int i_equal = threadIdx.y;
        int j_equal = threadIdx.x;
        i = k_tile*blockDim.y + threadIdx.y;
        j = blockIdx.x*blockDim.x + threadIdx.x;
        int index = i*n + j;
        local_KK_tile[i_equal][j_equal] = dist[index - (blockIdx.x - k_tile)*blockDim.x];
        local_KI_tile[i_equal][j_equal] = dist[index];
	__syncthreads();

        for(inner_k=0; inner_k<tile_dim; inner_k++){
            if( local_KI_tile[i_equal][j_equal] > local_KK_tile[i_equal][inner_k] + local_KI_tile[inner_k][j_equal] ){
                local_KI_tile[i_equal][j_equal] = local_KK_tile[i_equal][inner_k] + local_KI_tile[inner_k][j_equal];
            }
            __syncthreads();
        }

        dist[index] = local_KI_tile[i_equal][j_equal];
    }
}

__global__ void GPU_KERNEL_NAME(_tiled_shmem_stage_3)(weight_t *dist, int n,
                                                      int k_tile, int tile_dim)
{
    int i,j,i_equal,j_equal,inner_k;

    __shared__ weight_t local_IJ_tile[GPU_TILE_DIM][GPU_TILE_DIM];
    __shared__ weight_t local_IK_tile[GPU_TILE_DIM][GPU_TILE_DIM];
    __shared__ weight_t local_KJ_tile[GPU_TILE_DIM][GPU_TILE_DIM];

    i_equal = threadIdx.y;      
    j_equal = threadIdx.x;      //i_equal,j_equal indexes inside the tile
    i = blockIdx.y*blockDim.y + threadIdx.y;
    j = blockIdx.x*blockDim.x + threadIdx.x;        //i,j indexes in the whole array
    int index = i*n + j; 
    if( (blockIdx.x != k_tile) && (blockIdx.y != k_tile) ){       //this will be executed only by the blocks not belonging to the k_tile
        
        local_IJ_tile[i_equal][j_equal] = dist[index];
        local_IK_tile[i_equal][j_equal] = dist[index - (blockIdx.x - k_tile)*blockDim.x];
        local_KJ_tile[i_equal][j_equal] = dist[index - (blockIdx.y - k_tile)*blockDim.y*n];
	__syncthreads();

        for(inner_k=0; inner_k<tile_dim; inner_k++){
            if( local_IJ_tile[i_equal][j_equal] > local_IK_tile[i_equal][inner_k] + local_KJ_tile[inner_k][j_equal] ){
                local_IJ_tile[i_equal][j_equal] = local_IK_tile[i_equal][inner_k] + local_KJ_tile[inner_k][j_equal];
            }
        }

        dist[index] = local_IJ_tile[i_equal][j_equal];
    }

}


graph_t *MAKE_KERNEL_NAME(_gpu, _naive)(graph_t *graph)
{
    xtimer_t transfer_timer;
    timer_clear(&transfer_timer);
    timer_start(&transfer_timer);
    weight_t *dist_gpu = copy_graph_to_gpu(graph);
    timer_stop(&transfer_timer);

    /* FILLME: Set up and launch the kernel(s) */
    int block_dim = 32;
    int k;


    /* 1-Dimension Grid */
    //dim3 dimBlock(block_dim*block_dim);
    //dim3 dimGrid( (graph->nr_vertices/block_dim)*(graph->nr_vertices/block_dim) );
    //check 54    
    /* 2-Dimensions Grid */
    dim3 dimBlock(block_dim,block_dim);
    dim3 dimGrid( (graph->nr_vertices/block_dim),(graph->nr_vertices/block_dim) );


    for(k=0; k<graph->nr_vertices; k++){         //NOTICE: in CUDA the whole array is represented as 1D (nxn size)
        GPU_KERNEL_NAME(_naive) <<< dimGrid, dimBlock >>> (dist_gpu, graph->nr_vertices,k);         
        hipDeviceSynchronize();//wait for the last kernel to finish and then go to next k for right results
    }

    /*
     * Wait for last kernel to finish, so as to measure correctly the
     * transfer times Otherwise, copy from GPU will block
     */

    /* Copy back results to host */
    timer_start(&transfer_timer);
    copy_graph_from_gpu(dist_gpu, graph);
    timer_stop(&transfer_timer);
    printf("Total transfer times: %lf s\n",
           timer_elapsed_time(&transfer_timer));
    return graph;
}

graph_t *MAKE_KERNEL_NAME(_gpu, _tiled)(graph_t *graph)
{
    xtimer_t transfer_timer;
    timer_clear(&transfer_timer);
    timer_start(&transfer_timer);
    weight_t *dist_gpu = copy_graph_to_gpu(graph);
    timer_stop(&transfer_timer);

    int tile_dimension = GPU_TILE_DIM;
    int tiles_number_diagonally = graph->nr_vertices / tile_dimension;
    int k;
    //dim3 dimGrid(graph->nr_vertices/GPU_TILE_DIM,graph->nr_vertices/GPU_TILE_DIM);
    //dim3 dimBlock(GPU_TILE_DIM,GPU_TILE_DIM);

    /*
     * FILLME: Set up and launch the kernel(s)
     *
     * You may need different grid/block configurations for each stage
     * of the computation
     * 
     * Use GPU_TILE_DIM (see graph.h) as the tile dimension. You can
     * adjust its value during compilation. See `make help' for more
     * information.
     */
     for(k=0; k<tiles_number_diagonally; k++){
            /* --- PHASE 1 ---*/
	    dim3 dimGrid(1,1);
	    dim3 dimBlock(GPU_TILE_DIM,GPU_TILE_DIM);
            GPU_KERNEL_NAME(_tiled_stage_1) <<< dimGrid, dimBlock >>> (dist_gpu, graph->nr_vertices, k , tile_dimension);
            hipDeviceSynchronize();
            //printf("\n ***GPU****Stage 1<->k=%d*****GPU*****\n",k);
            //copy_graph_from_gpu(dist_gpu, graph);
            //graph_print(graph);


	    dim3 dimGrid2(graph->nr_vertices/GPU_TILE_DIM,2);
	    dim3 dimBlock2(GPU_TILE_DIM,GPU_TILE_DIM);
            /* --- PHASE 2 ---*/
            GPU_KERNEL_NAME(_tiled_stage_2) <<< dimGrid2, dimBlock2 >>> (dist_gpu, graph->nr_vertices, k , tile_dimension);
            hipDeviceSynchronize();
            //printf("\n ***GPU****Stage 2<->k=%d****GPU******\n",k);
            //copy_graph_from_gpu(dist_gpu, graph);
            //graph_print(graph);

            /* --- PHASE 3 ---*/
	    dim3 dimGrid3(graph->nr_vertices/GPU_TILE_DIM,graph->nr_vertices/GPU_TILE_DIM);
	    dim3 dimBlock3(GPU_TILE_DIM,GPU_TILE_DIM);
            GPU_KERNEL_NAME(_tiled_stage_3) <<< dimGrid3, dimBlock3 >>> (dist_gpu, graph->nr_vertices, k , tile_dimension);
            hipDeviceSynchronize();
            //printf("\n ****GPU*****Stage 3<->k=%d*****GPU*****\n",k);
            //copy_graph_from_gpu(dist_gpu, graph);
            //graph_print(graph);
     }


    /*
     * Wait for last kernel to finish, so as to measure correctly the
     * transfer times Otherwise, copy from GPU will block
     */
    hipDeviceSynchronize();

    /* Copy back results to host */
    timer_start(&transfer_timer);
    copy_graph_from_gpu(dist_gpu, graph);
    timer_stop(&transfer_timer);
    printf("Total transfer times: %lf s\n",
           timer_elapsed_time(&transfer_timer));
    return graph;
}

graph_t *MAKE_KERNEL_NAME(_gpu, _tiled_shmem)(graph_t *graph)
{
    xtimer_t transfer_timer;
    timer_clear(&transfer_timer);
    timer_start(&transfer_timer);
    weight_t *dist_gpu = copy_graph_to_gpu(graph);
    timer_stop(&transfer_timer);

    int tile_dimension = GPU_TILE_DIM;
    int tiles_number_diagonally = graph->nr_vertices / tile_dimension;
    int k;
    //dim3 dimGrid(graph->nr_vertices/GPU_TILE_DIM,graph->nr_vertices/GPU_TILE_DIM);
    //dim3 dimBlock(GPU_TILE_DIM,GPU_TILE_DIM);

    /*
     * FILLME: Set up and launch the kernel(s)
     *
     * You may need different grid/block configurations for each stage
     * of the computation
     * 
     * Use GPU_TILE_DIM (see graph.h) as the tile dimension. You can
     * adjust its value during compilation. See `make help' for more
     * information.
     */

     for(k=0; k<tiles_number_diagonally; k++){
            /* --- PHASE 1 ---*/
	    dim3 dimGrid(1,1);
	    dim3 dimBlock(GPU_TILE_DIM,GPU_TILE_DIM);
            GPU_KERNEL_NAME(_tiled_shmem_stage_1) <<< dimGrid, dimBlock  >>> (dist_gpu, graph->nr_vertices, k , tile_dimension);
            hipDeviceSynchronize();
            //printf("\n ***GPU****Stage 1<->k=%d*****GPU*****\n",k);
            //copy_graph_from_gpu(dist_gpu, graph);
            //graph_print(graph);

            /* --- PHASE 2 ---*/
	    dim3 dimGrid2(graph->nr_vertices/GPU_TILE_DIM,2);
	    dim3 dimBlock2(GPU_TILE_DIM,GPU_TILE_DIM);
            GPU_KERNEL_NAME(_tiled_shmem_stage_2) <<< dimGrid2, dimBlock2 >>> (dist_gpu, graph->nr_vertices, k , tile_dimension);
            hipDeviceSynchronize();
            //printf("\n ***GPU****Stage 2<->k=%d****GPU******\n",k);
            //copy_graph_from_gpu(dist_gpu, graph);
            //graph_print(graph);

            /* --- PHASE 3 ---*/
	    dim3 dimGrid3(graph->nr_vertices/GPU_TILE_DIM,graph->nr_vertices/GPU_TILE_DIM);
            dim3 dimBlock3(GPU_TILE_DIM,GPU_TILE_DIM);
            GPU_KERNEL_NAME(_tiled_shmem_stage_3) <<< dimGrid3, dimBlock3 >>> (dist_gpu, graph->nr_vertices, k , tile_dimension);
            hipDeviceSynchronize();
            //printf("\n ****GPU*****Stage 3<->k=%d*****GPU*****\n",k);
            //copy_graph_from_gpu(dist_gpu, graph);
            //graph_print(graph);
     }

    /*
     * Wait for last kernel to finish, so as to measure correctly the
     * transfer times Otherwise, copy from GPU will block
     */
    hipDeviceSynchronize();

    /* Copy back results to host */
    timer_start(&transfer_timer);
    copy_graph_from_gpu(dist_gpu, graph);
    timer_stop(&transfer_timer);
    printf("Total transfer times: %lf s\n",
           timer_elapsed_time(&transfer_timer));
    return graph;
}
